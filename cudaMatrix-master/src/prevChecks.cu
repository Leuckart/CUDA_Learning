#include "prevChecks.h"

#include "errorMacros.h"

void checkCLIArguments(int argc, char *argv[], unsigned int *width)
{
	unsigned int i, ovfCheck, curDigit;

	//basic syntax and positive integer
	if(argc!=2)
		SPIT("Usage: %s <matrix_width>\n", argv[0]);

	(*width) = 0;
	for(i=0; i<strlen(argv[1]); i++)
	{
		(*width) *= 10;
		if(argv[1][i]<'0' || argv[1][i]>'9')
			SPIT("Could not parse \"%s\" as positive integer\n", argv[1]);
		//sum with overflow check
		curDigit = (unsigned int) (argv[1][i]-'0');
		ovfCheck = (*width);
		(*width) += curDigit;
		if((*width) < curDigit | (*width) < ovfCheck)
			SPIT("Integer \"%s\" too big, overflows\n", argv[1]);
	}
}

void checkCUDAPresent(void)
{
	int count;
	hipGetDeviceCount(&count);
	if(count>0)
		return;
	SPIT("No CUDA capable devices found!\n");
}
