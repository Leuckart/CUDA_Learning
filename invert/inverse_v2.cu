#include "hip/hip_runtime.h"
/**************************************************
	> File Name:  invert.cpp
	> Author:     Leuckart
	> Time:       2018-12-09 19:13
**************************************************/

#include "inverse.h"

double Get_Det(double *mat, int n)
{
	if (n == 1)
	{
		return mat[0];
	}
	double ans = 0;
	double *cof = (double *)malloc((n - 1) * (n - 1) * sizeof(double));

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n - 1; j++)
		{
			for (int k = 0; k < n - 1; k++)
			{
				Point(cof, j, k, n - 1) = Point(mat, j + 1, k < i ? k : k + 1, n);
			}
		}
		double t = Get_Det(cof, n - 1);
		ans += mat[i] * t * (i % 2 == 0 ? 1 : -1);
	}
	free(cof);
	return ans;
}

__device__ double Loop(double *mat,int n)
{
	if (n == 1)
	{
		return mat[0];
	}
	double ans = 0;
	double *cof = (double *)malloc((n - 1) * (n - 1) * sizeof(double));

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n - 1; j++)
		{
			for (int k = 0; k < n - 1; k++)
			{
				Point(cof, j, k, n - 1) = Point(mat, j + 1, k < i ? k : k + 1, n);
			}
		}
		double t = Loop(cof, n - 1);
		ans += mat[i] * t * (i % 2 == 0 ? 1 : -1);
	}
	free(cof);
	return ans;
}

__global__ void Kernel_Function(double *ori,double *inv,double det)
{
	const unsigned int _idx=(blockIdx.x*blockDim.x)+threadIdx.x;
	const unsigned int _idy=(blockIdx.y*blockDim.y)+threadIdx.y;

	double *cof;
	cof=(double *)malloc((SIZE - 1) * (SIZE - 1) * sizeof(double));
	int i=_idy;
	int j=_idx;

	for (int k = 0; k < SIZE - 1; k++)
	{
		for (int t = 0; t < SIZE - 1; t++)
		{
			Point(cof, k, t, SIZE - 1) = Point(ori, k < i ? k : k + 1, t < j ? t : t + 1, SIZE);
		}
	}
	Point(inv, j, i, SIZE) = Loop(cof,SIZE-1)* ((i + j) % 2 == 0 ? 1 : -1)/det;
	free(cof);
}

void Inverse_Matrix_Handle(double *ori, double *inv,dim3 Blocks_Per_Grid,dim3 Threads_Per_Block,double det)
{
	if (0 == det)
	{
		cout << "Warning : Singular Matrix !" << endl;
		exit(1);
	}
	Kernel_Function<<<Blocks_Per_Grid,Threads_Per_Block>>>(ori,inv,det);
}

int main()
{
	unsigned int Byte_Size = SIZE * SIZE * sizeof(double);
	double *Matrix_Ori = (double *)malloc(Byte_Size);
	Initialize_Matrix(Matrix_Ori);
	Show_Matrix(Matrix_Ori, "Original Matrix :");

	double *Matrix_Inv = (double *)malloc(Byte_Size);

	/* Initial Threads Blocks Begin */
	int thread_xdim = SIZE;
	int thread_ydim = SIZE;
	const dim3 Threads_Per_Block(thread_xdim, thread_ydim);
	const dim3 Blocks_Per_Grid(1, 1);
	/* Initial Threads Blocks End */

	/* Initial Memory Begin */
	double *Matrix_GPU;
	double *Matrix_Inv_GPU;
	Cuda_Call(hipMalloc((void **)&Matrix_GPU, Byte_Size));
	Cuda_Call(hipMalloc((void **)&Matrix_Inv_GPU, Byte_Size));
	Cuda_Call(hipMemcpy(Matrix_GPU, Matrix_Ori, Byte_Size, hipMemcpyHostToDevice));
	/* Initial Memory Begin */

	/* Test On Every Device Begin */
	int Device_All;
	Cuda_Call(hipGetDeviceCount(&Device_All));
	/* Test On Every Device End */

	for (int device_number = 0; device_number < Device_All; ++device_number)
	{
		/* Set Device Parameters Begin */
		Cuda_Call(hipSetDevice(device_number));
		struct hipDeviceProp_t device_prop;
		char device_prefix[100];
		Cuda_Call(hipGetDeviceProperties(&device_prop, device_number));
		sprintf(device_prefix, "ID: %d %s: ", device_number, device_prop.name);
		/* Set Device Parameters End */

		/* Initial Time Block Begin */
		hipEvent_t kernel_start, kernel_stop;
		float delta_time = 0.;
		Cuda_Call(hipEventCreate(&kernel_start));
		Cuda_Call(hipEventCreateWithFlags(&kernel_stop, hipEventBlockingSync));
		Cuda_Call(hipEventRecord(kernel_start, 0));
		/* Initial Time Block End */

		/* Kernel Function Execute Begin */
		float det=Get_Det(Matrix_Ori,SIZE);
		Inverse_Matrix_Handle(Matrix_GPU,Matrix_Inv_GPU,Blocks_Per_Grid,Threads_Per_Block,det);
		Cuda_Call(hipMemcpy(Matrix_Inv, Matrix_Inv_GPU, Byte_Size, hipMemcpyDeviceToHost));
		/* Kernel Function Execute End */

		/* Time Clock Begin */
		Cuda_Call(hipEventRecord(kernel_stop, 0));
		Cuda_Call(hipEventSynchronize(kernel_stop));
		Cuda_Call(hipEventElapsedTime(&delta_time, kernel_start, kernel_stop));
		printf("%s %.5fms\n", device_prefix, delta_time);
		Cuda_Call(hipEventDestroy(kernel_start));
		Cuda_Call(hipEventDestroy(kernel_stop));
		/* Time Clock End */
	}

	//double *Matrix_Res = (double *)malloc(Byte_Size);
	//Matrix_Mult(Matrix_Ori, Matrix_Inv, Matrix_Res);
	//Show_Matrix(Matrix_Res, "Mult Matrix :");

	/* Free Memory Begin */
	Cuda_Call(hipFree(Matrix_GPU));
	free(Matrix_Ori);
	free(Matrix_Inv);
	/* Free Memory End */
	return 0;
}
