#include "hip/hip_runtime.h"
/**************************************************
	> File Name:  invert.cpp
	> Author:     Leuckart
	> Time:       2018-12-09 19:13
**************************************************/

#include "invert.h"

double Get_Det(double *mat, int n)
{
	if (n == 1)
	{
		return mat[0];
	}
	double ans = 0;
	double *cof = (double *)malloc((n - 1) * (n - 1) * sizeof(double));

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n - 1; j++)
		{
			for (int k = 0; k < n - 1; k++)
			{
				Point(cof, j, k, n - 1) = Point(mat, j + 1, k < i ? k : k + 1, n);
			}
		}
		double t = Get_Det(cof, n - 1);
		ans += mat[i] * t * (i % 2 == 0 ? 1 : -1);
	}
	free(cof);
	return ans;
}

void Inverse_Matrix(double *ori, double *inv)
{
	double det = Get_Det(ori, SIZE);
	if (0 == det)
	{
		cout << "Warning : Singular Matrix !" << endl;
		exit(1);
	}

	double *cof = (double *)malloc((SIZE - 1) * (SIZE - 1) * sizeof(double));
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			for (int k = 0; k < SIZE - 1; k++)
			{
				for (int t = 0; t < SIZE - 1; t++)
				{
					Point(cof, k, t, SIZE - 1) = Point(ori, k < i ? k : k + 1, t < j ? t : t + 1, SIZE);
				}
			}
			Point(inv, j, i, SIZE) = Get_Det(cof, SIZE - 1) * ((i + j) % 2 == 0 ? 1 : -1)/det;
		}
	}
	free(cof);
}

__global__ void Kernel_Function(double *ori,double *inv,int now)
{
	const unsigned int _idx=(blockIdx.x*blockDim.x)+threadIdx.x;
	const unsigned int _idy=(blockIdx.y*blockDim.y)+threadIdx.y;
	if(_idy==0)
	{
		return;
	}
	
	const unsigned int index=((gridDim.x*blockDim.x)*_idx)+_idy;

	__shared__ double memory[SIZE];
	if(_idy!=0)
	{
		memory[_idx]=Point(ori,_idx,_idx,SIZE);
	}
	__syncthreads();

	//inv[index]=index;
	//inv[_idx*SIZE+_idy]+=index;
	//inv[index]=ori[index];

	//__syncthreads();
	//__shared__ double ii;
	//double ii;
	//ii=Point(ori,now,now,SIZE);
	double ii=memory[now];
	double temp=0.0;

	/*__syncthreads();
	if(_idx==now)
	{
		temp=1./ii;//1./Point(ori,_idx,now,SIZE);
		for(int i=0;i<SIZE;i++)
		{
			Point(ori,now,i,SIZE)*=temp;
			Point(inv,now,i,SIZE)*=temp;
		}
	}
	__syncthreads();*/

	if(_idx!=now)
	{
		temp=Point(ori,_idx,now,SIZE)/ii;
		//temp=ii/Point(ori,_idx,now,SIZE);
		for(int i=0;i<SIZE;i++)
		{
			Point(ori,_idx,i,SIZE)-=Point(ori,now,i,SIZE)*temp;
			Point(inv,_idx,i,SIZE)-=Point(inv,now,i,SIZE)*temp;
		}
	}
	else
	{
		return;
	}
}

__global__ void Kernel_Normalize(double *ori,double *inv)
{
	const unsigned int _idx=(blockIdx.x*blockDim.x)+threadIdx.x;
	const unsigned int _idy=(blockIdx.y*blockDim.y)+threadIdx.y;
	if(_idy==0)
	{
		return;
	}
	
	const unsigned int index=((gridDim.x*blockDim.x)*_idx)+_idy;

	//__shared__ double ii;
	double ii;
	ii=Point(ori,_idx,_idx,SIZE);
	double temp=0.0;

	temp=1./ii;//1./Point(ori,_idx,now,SIZE);
	for(int i=0;i<SIZE;i++)
	{
		Point(ori,_idx,i,SIZE)*=temp;
		Point(inv,_idx,i,SIZE)*=temp;
	}
	//__syncthreads();
}

void Inverse_Matrix_Handle(double *ori, double *inv,dim3 Blocks_Per_Grid,dim3 Threads_Per_Block)
{
	for(int i=0;i<SIZE;i++)
	{
		Kernel_Function<<<Blocks_Per_Grid,Threads_Per_Block>>>(ori,inv,i);
		hipDeviceSynchronize();
		//Kernel_Normalize<<<Blocks_Per_Grid,Threads_Per_Block>>>(ori,inv,i);
		//hipDeviceSynchronize();
	}
	Kernel_Normalize<<<Blocks_Per_Grid,Threads_Per_Block>>>(ori,inv);
	hipDeviceSynchronize();
}

void Show_Matrix(double *mat, const char *mesg)
{
	cout << mesg << endl;

	unsigned int mat_size = SIZE * SIZE;
	int flag = 0;
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			if(Point(mat,i,j,SIZE)<0.00001&&Point(mat,i,j,SIZE)>-0.00001)
			{
				cout<<"0"<<" ";
				//cout << Point(mat, i, j, SIZE) << " ";
			}
			else
			{
				cout << Point(mat, i, j, SIZE) << " ";
			}
		}
		cout << endl;
	}
	cout << endl;
}

void Initialize_Matrix(double *mat)
{
	/* should replace by urandom. Leuckart. */
	srand((unsigned)time(0));
	unsigned int mat_size = SIZE * SIZE;

	for (int i = 0; i < mat_size; i++)
	{
		mat[i] = rand() % 100 * 0.01;
	}
}

int main()
{
	unsigned int Byte_Size = SIZE * SIZE * sizeof(double);
	double *Matrix_Ori = (double *)malloc(Byte_Size);

	Initialize_Matrix(Matrix_Ori);
	Show_Matrix(Matrix_Ori, "Original Matrix :");

	//cout << Get_Det(Matrix_Ori, SIZE) << endl;

	double *Matrix_Inv = (double *)malloc(Byte_Size);
	Inverse_Matrix(Matrix_Ori, Matrix_Inv);
	Show_Matrix(Matrix_Inv, "Inverse Matrix :");

	double *Matrix_Inv_Inv = (double *)malloc(Byte_Size);
	//Inverse_Matrix(Matrix_Inv, Matrix_Inv_Inv);
	//Show_Matrix(Matrix_Inv_Inv, "Inverse Inverse Matrix :");

	/* Initial Threads Blocks Begin */
	int thread_xdim = SIZE;
	int thread_ydim = SIZE;
	const dim3 Threads_Per_Block(thread_xdim, thread_ydim);
	const dim3 Blocks_Per_Grid(1, 1);
	/* Initial Threads Blocks End */

	/* Initial Memory Begin */
	double *Matrix_GPU;
	double *Matrix_Inv_GPU;
	double *Matrix_Inv_Inv_GPU;
	double *ident=(double *)malloc(Byte_Size);
	for(int i=0;i<SIZE;i++)
	{
		Point(ident,i,i,SIZE)=1;
	}
	Cuda_Call(hipMalloc((void **)&Matrix_GPU, Byte_Size));
	Cuda_Call(hipMalloc((void **)&Matrix_Inv_GPU, Byte_Size));
	Cuda_Call(hipMalloc((void **)&Matrix_Inv_Inv_GPU, Byte_Size));
	Cuda_Call(hipMemcpy(Matrix_GPU, Matrix_Ori, Byte_Size, hipMemcpyHostToDevice));
	Cuda_Call(hipMemcpy(Matrix_Inv_GPU, ident, Byte_Size, hipMemcpyHostToDevice));
	Cuda_Call(hipMemcpy(Matrix_Inv_Inv_GPU, ident, Byte_Size, hipMemcpyHostToDevice));
	/* Initial Memory Begin */

	/* Test On Every Device Begin */
	int Device_All;
	Cuda_Call(hipGetDeviceCount(&Device_All));
	/* Test On Every Device End */

	for (int device_number = 0; device_number < Device_All; ++device_number)
	{
		/* Set Device Parameters Begin */
		Cuda_Call(hipSetDevice(device_number));
		struct hipDeviceProp_t device_prop;
		char device_prefix[100];
		Cuda_Call(hipGetDeviceProperties(&device_prop, device_number));
		sprintf(device_prefix, "ID: %d %s: ", device_number, device_prop.name);
		/* Set Device Parameters End */

		/* Initial Time Block Begin */
		hipEvent_t kernel_start, kernel_stop;
		float delta_time = 0.;
		Cuda_Call(hipEventCreate(&kernel_start));
		Cuda_Call(hipEventCreateWithFlags(&kernel_stop, hipEventBlockingSync));
		Cuda_Call(hipEventRecord(kernel_start, 0));
		/* Initial Time Block End */

		/* Kernel Function Execute Begin */
		Inverse_Matrix_Handle(Matrix_GPU,Matrix_Inv_GPU,Blocks_Per_Grid,Threads_Per_Block);
		//Inverse_Matrix_Handle(Matrix_Inv_GPU,Matrix_Inv_Inv_GPU,Blocks_Per_Grid,Threads_Per_Block);
		//Inverse_Matrix(Matrix_Ori,Matrix_Inv);
		/* Kernel Function Execute End */

		/* Time Clock Begin */
		Cuda_Call(hipEventRecord(kernel_stop, 0));
		Cuda_Call(hipEventSynchronize(kernel_stop));
		Cuda_Call(hipEventElapsedTime(&delta_time, kernel_start, kernel_stop));
		printf("%s %.5fms\n", device_prefix, delta_time);
		Cuda_Call(hipEventDestroy(kernel_start));
		Cuda_Call(hipEventDestroy(kernel_stop));
		/* Time Clock End */
	}

	/* Copy GPU To CPU Begin */
	Cuda_Call(hipMemcpy(Matrix_Inv, Matrix_Inv_GPU, Byte_Size, hipMemcpyDeviceToHost));
	//Cuda_Call(hipMemcpy(Matrix_Ori, Matrix_GPU, Byte_Size, hipMemcpyDeviceToHost));
	Cuda_Call(hipMemcpy(Matrix_Inv_Inv, Matrix_Inv_Inv_GPU, Byte_Size, hipMemcpyDeviceToHost));
	/* Copy GPU To CPU End */

	Show_Matrix(Matrix_Ori, "Original Matrix (Should Be I):");
	Show_Matrix(Matrix_Inv, "Inv Matrix :");
	//Show_Matrix(Matrix_Inv_Inv, "Inv Inv Matrix :");

	double *Matrix_Mult = (double *)malloc(Byte_Size);
	for(int i=0;i<SIZE;i++)
	{
		for(int j=0;j<SIZE;j++)
		{
			double temp=0.0;
			for(int k=0;k<SIZE;k++)
			{
				temp+=Point(Matrix_Ori,i,k,SIZE)*Point(Matrix_Inv,k,j,SIZE);
			}
			Point(Matrix_Mult,i,j,SIZE)=temp;
		}
	}
	Show_Matrix(Matrix_Mult, "Mult Matrix :");

	/* Free Memory Begin */
	Cuda_Call(hipFree(Matrix_GPU));
	free(Matrix_Ori);
	/* Free Memory End */
	return 0;
}
