#include "hip/hip_runtime.h"
/**************************************************
	> File Name:  invert.cpp
	> Author:     Leuckart
	> Time:       2018-12-09 19:13
**************************************************/

#include "invert.h"

float Get_Det(float *mat, int n)
{
	if (n == 1)
	{
		return mat[0];
	}
	float ans = 0;
	float *temp = (float *)malloc((n-1) * (n-1) * sizeof(float));

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n - 1; j++)
		{
			for (int k = 0; k < n - 1; k++)
			{
				Point(temp,j,k,n-1)=Point(mat,j+1,k<i?k:k+1,n);
			}
		}
		float t = Get_Det(temp, n - 1);
		ans += mat[i] * t * (i % 2 == 0 ? 1 : -1);
	}
	free(temp);
	return ans;
}

void Get_Adj(float *arcs, float *ans)
{
	if (SIZE == 1)
	{
		ans[0] = 1;
		return;
	}
	float *temp = (float *)malloc((SIZE-1) * (SIZE-1) * sizeof(float));
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			for (int k = 0; k < SIZE - 1; k++)
			{
				for (int t = 0; t < SIZE - 1; t++)
				{
					Point(temp,k,t,SIZE-1)=Point(arcs,k<i?k:k+1,t<j?t:t+1,SIZE);
				}
			}

			Point(ans, j, i, SIZE) = Get_Det(temp, SIZE - 1);
			if ((i + j) % 2 == 1)
			{
				Point(ans, j, i, SIZE) = -Point(ans, j, i, SIZE);
			}
		}
	}
	free(temp);
}

void Inverse_Matrix(float *src, float *des)
{
	float flag = Get_Det(src, SIZE);
	float *t = (float *)malloc(SIZE * SIZE * sizeof(float));
	if (0 == flag)
	{
		cout << "Warning : Singular Matrix !" << endl;
		exit(1);
	}
	else
	{
		Get_Adj(src, t);

		for (int i = 0; i < SIZE; i++)
		{
			for (int j = 0; j < SIZE; j++)
			{
				Point(des, i, j, SIZE) = Point(t, i, j, SIZE) / flag;
			}
		}
	}
	free(t);
}

void Show_Matrix(float *mat, const char *mesg)
{
	cout << mesg << endl;

	unsigned int mat_size = SIZE * SIZE;
	int flag = 0;
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			cout << Point(mat, i, j, SIZE) << " ";
		}
		cout << endl;
	}
	cout << endl;
}

void Initialize_Matrix(float *mat)
{
	/* should replace by urandom. Leuckart. */
	srand((unsigned)time(0));
	unsigned int mat_size = SIZE * SIZE;

	for (int i = 0; i < mat_size; i++)
	{
		mat[i] = rand() % 100 * 0.01;
	}
}

__global__ void Matrix_Mult(float mata[SIZE][SIZE], float matb[SIZE][SIZE])
{
	//print()
	//cout<<mata[0][0]<<" "<<matb[0][0]<<endl;
}

int main()
{
	unsigned int Byte_Size = SIZE * SIZE * sizeof(float);
	float *Matrix_Ori = (float *)malloc(Byte_Size);

	Initialize_Matrix(Matrix_Ori);
	Show_Matrix(Matrix_Ori, "Original Matrix :");

	cout<<Get_Det(Matrix_Ori,SIZE)<<endl;

	float *Matrix_Inv = (float *)malloc(Byte_Size);
	Inverse_Matrix(Matrix_Ori, Matrix_Inv);
	Show_Matrix(Matrix_Inv, "Inverse Matrix :");

	float *Matrix_Inv_Inv = (float *)malloc(Byte_Size);
	Inverse_Matrix(Matrix_Inv, Matrix_Inv_Inv);
	Show_Matrix(Matrix_Inv_Inv, "Inverse Inverse Matrix :");

	/* Initial Threads Blocks Begin */
	//int thread_xdim = SIZE;
	//int thread_ydim = SIZE;
	//const dim3 Threads_Per_Block(thread_xdim, thread_ydim);
	//const dim3 Blocks_Per_Grid(1, 1);
	/* Initial Threads Blocks End */

	/* Initial Memory Begin */
	//float *Matrix_CPU=original_matrix;//(float *)malloc(Byte_Size);
	//float *Matrix_GPU;
	//Cuda_Call(hipMalloc((void **)&Matrix_GPU,Byte_Size));
	/* Initial Memory Begin */

	free(Matrix_Ori);
	//free(Matrix_Inv);
	//free(Matrix_Inv_Inv);
	return 0;
}
