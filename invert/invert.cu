#include "hip/hip_runtime.h"
/**************************************************
	> File Name:  invert.cpp
	> Author:     Leuckart
	> Time:       2018-12-09 19:13
**************************************************/

#include "invert.h"

double Get_Det(double *mat, int n)
{
	if (n == 1)
	{
		return mat[0];
	}
	double ans = 0;
	double *cof = (double *)malloc((n - 1) * (n - 1) * sizeof(double));

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n - 1; j++)
		{
			for (int k = 0; k < n - 1; k++)
			{
				Point(cof, j, k, n - 1) = Point(mat, j + 1, k < i ? k : k + 1, n);
			}
		}
		double t = Get_Det(cof, n - 1);
		ans += mat[i] * t * (i % 2 == 0 ? 1 : -1);
	}
	free(cof);
	return ans;
}

void Inverse_Matrix(double *ori, double *inv)
{
	double det = Get_Det(ori, SIZE);
	if (0 == det)
	{
		cout << "Warning : Singular Matrix !" << endl;
		exit(1);
	}

	double *cof = (double *)malloc((SIZE - 1) * (SIZE - 1) * sizeof(double));
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			for (int k = 0; k < SIZE - 1; k++)
			{
				for (int t = 0; t < SIZE - 1; t++)
				{
					Point(cof, k, t, SIZE - 1) = Point(ori, k < i ? k : k + 1, t < j ? t : t + 1, SIZE);
				}
			}
			Point(inv, j, i, SIZE) = Get_Det(cof, SIZE - 1) * ((i + j) % 2 == 0 ? 1 : -1) / det;
		}
	}
	free(cof);
}

__global__ void Row_Kernel_Function(double *ori, double *inv, int now)
{
	const unsigned int _idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int _idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	const unsigned int thread_idx = ((gridDim.x * blockDim.x) * _idy) + _idx;

	const unsigned int idx = thread_idx / SIZE;
	const unsigned int idy = thread_idx % SIZE;

	if ((idx < SIZE) && (idy < SIZE))
	{
		double ii = Element(ori, now, now);
		double temp = 0.0;

		if (idx != now)
		{
			temp = Element(ori, idx, now) / ii;
			Element(ori, idx, idy) -= Element(ori, now, idy) * temp;
			Element(inv, idx, idy) -= Element(inv, now, idy) * temp;
		}
	}
	__syncthreads();

	/*
	__shared__ double memory[SIZE];
	if((idx<SIZE)&&(idy<SIZE)&&(idy==0))
	{
		memory[idx]=Element(ori,idx,idx);
	}
	__syncthreads();

	if((idx<SIZE)&&(idy<SIZE))
	{
		double ii=memory[now];
		double temp=0.0;

		if(idx!=now)
		{
			temp=Element(ori,idx,now)/ii;
			Element(ori,idx,idy)-=Element(ori,now,idy)*temp;
			Element(inv,idx,idy)-=Element(inv,now,idy)*temp;
		}
	}
	__syncthreads();
	*/
}

__global__ void Row_Kernel_Normalize(double *ori, double *inv)
{
	const unsigned int _idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int _idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	const unsigned int thread_idx = ((gridDim.x * blockDim.x) * _idy) + _idx;

	const unsigned int idx = thread_idx / SIZE;
	const unsigned int idy = thread_idx % SIZE;

	if ((idx < SIZE) && (idy < SIZE))
	{

		double temp = 1. / Element(ori, idx, idx);
		Element(ori, idx, idy) *= temp;
		Element(inv, idx, idy) *= temp;
	}
	__syncthreads();
	/*
	__shared__ double head[SIZE];
	if(idy==0)
	{
		head[idx]=1./Element(ori,idx,idx);
	}
	__syncthreads();

	Element(ori,idx,idy)*=head[idx];
	Element(inv,idx,idy)*=head[idx];
	*/
	//__syncthreads();
}

void Row_Function(double *ori, double *inv, int now)
{
	double ii = Element(ori, now, now);
	double temp = 0.0;
	for (int i = 0; i < SIZE; i++)
	{
		if (i == now)
		{
			continue;
		}
		temp = Element(ori, i, now) / ii;
		for (int j = 0; j < SIZE; j++)
		{
			Element(ori, i, j) -= Element(ori, now, j) * temp;
			Element(inv, i, j) -= Element(inv, now, j) * temp;
		}
	}
}

void Row_Normalize(double *ori, double *inv)
{
	for (int i = 0; i < SIZE; i++)
	{
		double temp = 1. / Element(ori, i, i);
		for (int j = 0; j < SIZE; j++)
		{
			//Element(ori,i,j)*=temp;
			Element(inv, i, j) *= temp;
		}
	}
}

void Inverse_Matrix_Handle(double *ori, double *inv)
{

	for (int i = 0; i < SIZE; i++)
	{
		Row_Function(ori, inv, i);
	}
	Row_Normalize(ori, inv);
}

void Inverse_Matrix_Kernel_Handle(double *ori, double *inv, dim3 Blocks_Per_Grid, dim3 Threads_Per_Block)
{

	for (int i = 0; i < SIZE; i++)
	{
		Row_Kernel_Function<<<Blocks_Per_Grid, Threads_Per_Block>>>(ori, inv, i);
		hipDeviceSynchronize();
	}
	Row_Kernel_Normalize<<<Blocks_Per_Grid, Threads_Per_Block>>>(ori, inv);
	hipDeviceSynchronize();
}

void Show_Matrix(double *mat, const char *mesg)
{
	cout << mesg << endl;
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			if (Element(mat, i, j) < 0.00001 && Element(mat, i, j) > -0.00001)
			{
				cout << "0"
					 << " ";
				//cout << Element(mat, i, j) << " ";
			}
			else
			{
				cout << Element(mat, i, j) << " ";
			}
		}
		cout << endl;
	}
	cout << endl;
}

void Initialize_Matrix(double *mat)
{
	/* should replace by urandom. Leuckart. */
	srand((unsigned)time(0));
	unsigned int mat_size = SIZE * SIZE;

	for (int i = 0; i < mat_size; i++)
	{
		mat[i] = rand() % 100 * 0.01;
	}
}

void Matrix_Mult(double *a, double *b, double *res)
{
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			double temp = 0.0;
			for (int k = 0; k < SIZE; k++)
			{
				temp += Element(a, i, k) * Element(b, k, j);
			}
			Element(res, i, j) = temp;
		}
	}
}

int main()
{
	unsigned int Byte_Size = SIZE * SIZE * sizeof(double);
	double *Matrix_Ori = (double *)malloc(Byte_Size);

	Initialize_Matrix(Matrix_Ori);
	//Show_Matrix(Matrix_Ori, "Original Matrix :");
	//cout << Get_Det(Matrix_Ori, SIZE) << endl;

	double *Matrix_Inv = (double *)malloc(Byte_Size);
	//Inverse_Matrix(Matrix_Ori, Matrix_Inv);
	//Show_Matrix(Matrix_Inv, "Inverse Matrix :");

	/* Initial Threads Blocks Begin */
	int thread_xdim = 32;
	int thread_ydim = 32;
	const dim3 Threads_Per_Block(thread_xdim, thread_ydim);
	const dim3 Blocks_Per_Grid(int((SIZE - 1) / Threads_Per_Block.x) + 1, int((SIZE - 1) / Threads_Per_Block.y) + 1);
	/* Initial Threads Blocks End */

	/* Initial Memory Begin */
	double *Matrix_GPU;
	double *Matrix_Inv_GPU;
	double *ident = (double *)malloc(Byte_Size);
	for (int i = 0; i < SIZE; i++)
	{
		Element(ident, i, i) = 1;
	}
	Cuda_Call(hipMalloc((void **)&Matrix_GPU, Byte_Size));
	//Cuda_Call(hipMalloc((void **)&Matrix_Inv_Inv_GPU, Byte_Size));
	Cuda_Call(hipMalloc((void **)&Matrix_Inv_GPU, Byte_Size));
	//Cuda_Call(hipMalloc((void **)&Matrix_Inv_Inv_GPU, Byte_Size));
	/* Initial Memory Begin */

	/* Test On Every Device Begin */
	int Device_All;
	Cuda_Call(hipGetDeviceCount(&Device_All));
	/* Test On Every Device End */

	for (int device_number = 0; device_number < Device_All; ++device_number)
	{
		/* Set Device Parameters Begin */
		Cuda_Call(hipSetDevice(device_number));
		struct hipDeviceProp_t device_prop;
		char device_prefix[100];
		Cuda_Call(hipGetDeviceProperties(&device_prop, device_number));
		sprintf(device_prefix, "ID: %d %s: ", device_number, device_prop.name);
		/* Set Device Parameters End */

		/* Initial Time Block Begin */
		hipEvent_t kernel_start, kernel_stop;
		float delta_time = 0.;
		Cuda_Call(hipEventCreate(&kernel_start));
		Cuda_Call(hipEventCreateWithFlags(&kernel_stop, hipEventBlockingSync));
		Cuda_Call(hipEventRecord(kernel_start, 0));
		/* Initial Time Block End */

		/* Kernel Function Execute Begin */
		Cuda_Call(hipMemcpy(Matrix_GPU, Matrix_Ori, Byte_Size, hipMemcpyHostToDevice));
		Cuda_Call(hipMemcpy(Matrix_Inv_GPU, ident, Byte_Size, hipMemcpyHostToDevice));
		Inverse_Matrix_Kernel_Handle(Matrix_GPU, Matrix_Inv_GPU, Blocks_Per_Grid, Threads_Per_Block);
		Cuda_Call(hipMemcpy(Matrix_Inv, Matrix_Inv_GPU, Byte_Size, hipMemcpyDeviceToHost));
		//Show_Matrix(Matrix_Inv, "");

		//double *Matrix_Ori_Copy = (double *)malloc(Byte_Size);
		//memcpy(Matrix_Ori_Copy, Matrix_Ori, Byte_Size);
		//Inverse_Matrix_Handle(Matrix_Ori_Copy, ident);
		//Show_Matrix(ident, "");

		/* Kernel Function Execute End */

		/* Time Clock Begin */
		Cuda_Call(hipEventRecord(kernel_stop, 0));
		Cuda_Call(hipEventSynchronize(kernel_stop));
		Cuda_Call(hipEventElapsedTime(&delta_time, kernel_start, kernel_stop));
		printf("%s %.5fms\n", device_prefix, delta_time);
		Cuda_Call(hipEventDestroy(kernel_start));
		Cuda_Call(hipEventDestroy(kernel_stop));
		/* Time Clock End */
	}

	/* Copy GPU To CPU Begin */
	//Cuda_Call(hipMemcpy(Matrix_Ori, Matrix_GPU, Byte_Size, hipMemcpyDeviceToHost));
	//Show_Matrix(Matrix_Ori,"...");
	/* Copy GPU To CPU End */

	double *Matrix_Res = (double *)malloc(Byte_Size);
	Matrix_Mult(Matrix_Ori, Matrix_Inv, Matrix_Res);
	////Matrix_Mult(Matrix_Ori,ident,Matrix_Res);
	//Show_Matrix(Matrix_Res, "Mult Matrix :");

	/* Free Memory Begin */
	Cuda_Call(hipFree(Matrix_GPU));
	free(Matrix_Ori);
	//free(Matrix_Res);
	/* Free Memory End */
	return 0;
}
